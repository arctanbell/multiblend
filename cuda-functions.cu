#include "hip/hip_runtime.h"
#include "cuda-functions.h"

#include <hip/hip_runtime.h>
#include <opencv2/cudaarithm.hpp>

//helpers
static int calc_drid_dim(int array_size, int block_size)
{
	return (array_size - 1) / block_size + 1;
}

__global__ void kernel_find_distances_cycle_y_horiz(cv::cuda::PtrStep<float> ptr_dist, cv::cuda::PtrStep<uint8_t> ptr_mat, cv::cuda::PtrStep<uint8_t> ptr_mask, int shift, int ybeg, int size, int xbeg, int xend, int l_straight)
{
	size_t y = blockIdx.x * blockDim.x + threadIdx.x + ybeg;
	
	if (y >= ybeg + size)
		return;

	auto pdist = ptr_dist.ptr(y);
	auto pmat = ptr_mat.ptr(y);
	auto pmask = ptr_mask.ptr(y);

	int x = xbeg;
	while (x != xend)
	{
		if (pmask[x])
		{
			x += shift;
			continue;
		}
		if (pdist[x - shift] + l_straight < pdist[x])
		{
			pdist[x] = pdist[x - shift] + l_straight;
			pmat[x] = pmat[x - shift];
		}
		x += shift;
	}
}

__global__ void kernel_find_distances_cycle_x(float *pdist, uint8_t *pnums, const uint8_t *pmask, const float *pdist_prev, const uint8_t *pnums_prev, int l_straight, int l_diag, int size)
{
	size_t x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x >= size || pmask[x] || pdist[x] == 0)
		return;

	if (pdist_prev[x] + l_straight < pdist[x])
	{
		pdist[x] = pdist_prev[x] + l_straight;
		pnums[x] = pnums_prev[x];
	}

	if (x != 0)
	{
		if (pdist_prev[x - 1] + l_diag < pdist[x])
		{
			pdist[x] = pdist_prev[x - 1] + l_diag;
			pnums[x] = pnums_prev[x - 1];
		}
	}

	if (x != (size - 1))
	{
		if (pdist_prev[x + 1] + l_diag < pdist[x])
		{
			pdist[x] = pdist_prev[x + 1] + l_diag;
			pnums[x] = pnums_prev[x + 1];
		}
	}
}

__global__ void kernel_init_seamdist(cv::cuda::PtrStepSz<int> ptr_dist, cv::cuda::PtrStep<uint8_t> ptr_nums, cv::cuda::PtrStep<uint8_t> ptr_outmask, cv::cuda::PtrStep<uint8_t> *ptr_masks, int size_masks)
{
	size_t pt = blockIdx.x * blockDim.x + threadIdx.x;
	if (pt >= ptr_dist.cols * ptr_dist.rows)
		return;
	int y = pt / ptr_dist.cols;
	int x = pt - ptr_dist.cols * y;

	int dist_max = 2 * (ptr_dist.cols + ptr_dist.rows);

	int* pdist = ptr_dist.ptr(y);
	uint8_t* pnums = ptr_nums.ptr(y);
	uint8_t* poutmask = ptr_outmask.ptr(y);

	int count = 0;
	int num = 0;
	for (int i = 0; i < size_masks; ++i)
		if (ptr_masks[i].ptr(y)[x]) //1-visible
		{
			++count;
			num = i;
		}

	if (count == 0)
		poutmask[x] = 1;
	else
		poutmask[x] = 0;

	if (count == 1)
	{
		pdist[x] = 0;
		pnums[x] = num;
	}
	else
	{
		pdist[x] = dist_max;
		pnums[x] = 0;
	}
}

__global__ void kernel_find_seamdistances_cycle_x(const uint8_t **pmasks, int numimages, const uint8_t *poutmask, int *pdist, const int *pdist_prev, uint8_t *pnums, const uint8_t *pnums_prev, int l_straight, int l_diag, int size)
{
	size_t x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x >= size || pdist[x] == 0)
		return;

	if (pdist_prev[x] + l_straight < pdist[x] && (pmasks[pnums_prev[x]][x] || poutmask[x]))
	{
		pdist[x] = pdist_prev[x] + l_straight;
		pnums[x] = pnums_prev[x];
	}

	if (x != 0)
	{
		if (pdist_prev[x - 1] + l_diag < pdist[x] && (pmasks[pnums_prev[x - 1]][x] || poutmask[x]))
		{
			pdist[x] = pdist_prev[x - 1] + l_diag;
			pnums[x] = pnums_prev[x - 1];
		}
	}

	if (x != (size - 1))
	{
		if (pdist_prev[x + 1] + l_diag < pdist[x] && (pmasks[pnums_prev[x + 1]][x] || poutmask[x]))
		{
			pdist[x] = pdist_prev[x + 1] + l_diag;
			pnums[x] = pnums_prev[x + 1];
		}
	}

}

__global__ void kernel_find_seamdistances_cycle_y_horiz(cv::cuda::PtrStep<int> ptr_dist, cv::cuda::PtrStep<uint8_t> ptr_mat, cv::cuda::PtrStep<uint8_t> ptr_outmask, cv::cuda::PtrStep<uint8_t> *ptr_masks, int shift, int ybeg, int size, int xbeg, int xend, int l_straight)
{
	size_t y = blockIdx.x * blockDim.x + threadIdx.x + ybeg;
	if (y >= ybeg + size)
		return;

	auto pdist = ptr_dist.ptr(y);
	auto pmat = ptr_mat.ptr(y);
	auto poutmask = ptr_outmask.ptr(y);
		
	int x = xbeg;
	while (x != xend)
	{
		if (pdist[x] == 0)
		{
			x += shift;
			continue;
		}
		if (pdist[x - shift] + l_straight < pdist[x] && (ptr_masks[pmat[x - shift]].ptr(y)[x] || poutmask[x]))
		{
			pdist[x] = pdist[x - shift] + l_straight;
			pmat[x] = pmat[x - shift];
		}
		x += shift;
	}
}

__global__ void kernel_extract_masks(cv::cuda::PtrStepSz<uint8_t> *ptr_masks, cv::cuda::PtrStepSz<uint8_t> ptr_cvseams, int mask_value)
{
	size_t pt = blockIdx.x * blockDim.x + threadIdx.x;
	if (pt >= ptr_cvseams.cols * ptr_cvseams.rows)
		return;
	int y = pt / ptr_cvseams.cols;
	int x = pt - ptr_cvseams.cols * y;

	auto pseam = ptr_cvseams.ptr(y);
	ptr_masks[pseam[x]].ptr(y)[x] = mask_value;
}

void cuda_find_distances_cycle_y_horiz(
	cv::cuda::GpuMat &dist, cv::cuda::GpuMat &mat, const cv::cuda::GpuMat &mask,
	int shift, int ybeg, int yend, int xbeg, int xend,
	int l_straight)
{
	int size = yend - ybeg;
	if (size < 1)
		return;

	int nthreads = 256;
	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(calc_drid_dim(size, block_dim.x * block_dim.y), 1);

	cv::cuda::PtrStep<float> ptr_dist = dist;
	cv::cuda::PtrStep<uint8_t> ptr_mat = mat;
	cv::cuda::PtrStep<uint8_t> ptr_mask = mask;

	/*hipEvent_t start, kernel;
	hipEventCreate(&start);
	hipEventCreate(&kernel);
	hipEventRecord(start, 0);*/
	kernel_find_distances_cycle_y_horiz <<<grid_dim, block_dim >>>(ptr_dist, ptr_mat, ptr_mask, shift, ybeg, size, xbeg, xend, l_straight);
	/*hipEventRecord(kernel, 0);
	hipEventSynchronize(kernel);

	float time_kernel;
	hipEventElapsedTime(&time_kernel, start, kernel);
	printf("cuda time_kernel: %f ms\n", time_kernel);*/
}

void cuda_find_distances_cycle_x(
	const uint8_t *pmask, float *pdist, const float *pdist_prev, uint8_t *pnums, const uint8_t *pnums_prev,
	int tmp_xbeg, int tmp_xend,
	int l_straight, int l_diag)
{
	int size = tmp_xend - tmp_xbeg;
	if (size < 1)
		return;

	int nthreads = 256;
	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(calc_drid_dim(size, block_dim.x * block_dim.y), 1);
	
	float *pdist_beg = pdist + tmp_xbeg;
	uint8_t *pnums_beg = pnums + tmp_xbeg;
	const uint8_t *pmask_beg = pmask + tmp_xbeg;
	const float *pdist_prev_beg = pdist_prev + tmp_xbeg;
	const uint8_t *pnums_prev_beg = pnums_prev + tmp_xbeg;

	/*hipEvent_t start, kernel;
	hipEventCreate(&start);
	hipEventCreate(&kernel);
	hipEventRecord(start, 0);*/
	kernel_find_distances_cycle_x<<<grid_dim, block_dim>>>(pdist_beg, pnums_beg, pmask_beg, pdist_prev_beg, pnums_prev_beg, l_straight, l_diag, size);
	/*hipEventRecord(kernel, 0);
	hipEventSynchronize(kernel);

	float time_kernel;
	hipEventElapsedTime(&time_kernel, start, kernel);
	printf("cuda time_kernel: %f ms\n", time_kernel);*/
}

void cuda_init_seamdist(cv::cuda::GpuMat &dist, cv::cuda::GpuMat &nums, cv::cuda::GpuMat &outmask, const std::vector<cv::cuda::GpuMat> &masks)
{
	int size = dist.cols * dist.rows;
	if (size < 1)
		return;

	int nthreads = 256;
	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(calc_drid_dim(size, block_dim.x * block_dim.y), 1);

	cv::cuda::PtrStepSz<int> ptr_dist = dist;
	cv::cuda::PtrStep<uint8_t> ptr_nums = nums;
	cv::cuda::PtrStep<uint8_t> ptr_outmask = outmask;
	int numimages = masks.size();
	std::vector<cv::cuda::PtrStep<uint8_t> > ptr_masks(numimages);
	for (int i = 0; i < numimages; ++i)
		ptr_masks[i] = masks[i];

	cv::cuda::PtrStep<uint8_t> *dev_ptr_masks;
	int bufsize = numimages * sizeof(cv::cuda::PtrStep<uint8_t>);
	hipMalloc(&dev_ptr_masks, bufsize);
	hipMemcpy(dev_ptr_masks, &ptr_masks[0], bufsize, hipMemcpyHostToDevice);

	/*hipEvent_t start, kernel;
	hipEventCreate(&start);
	hipEventCreate(&kernel);
	hipEventRecord(start, 0);*/
	kernel_init_seamdist<<<grid_dim, block_dim>>>(ptr_dist, ptr_nums, ptr_outmask, dev_ptr_masks, numimages);
	/*hipEventRecord(kernel, 0);
	hipEventSynchronize(kernel);

	float time_kernel;
	hipEventElapsedTime(&time_kernel, start, kernel);
	printf("cuda time_kernel: %f ms\n", time_kernel);*/
	hipFree(dev_ptr_masks);
}

void cuda_find_seamdistances_cycle_x(
	const std::vector<const uint8_t*> &pmasks, const uint8_t *poutmask, int *pdist, const int *pdist_prev, uint8_t *pnums, const uint8_t *pnums_prev,
	int tmp_xbeg, int tmp_xend,
	int l_straight, int l_diag)
{
	int size = tmp_xend - tmp_xbeg;
	if (size < 1)
		return;

	int nthreads = 256;
	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(calc_drid_dim(size, block_dim.x * block_dim.y), 1);

	int numimages = pmasks.size();
	std::vector<const uint8_t *> pmasks_beg(numimages);
	for (int i = 0; i < numimages; ++i)
		pmasks_beg[i] = pmasks[i] + tmp_xbeg;
	const uint8_t **dev_pmasks_beg;
	int bufsize = numimages * sizeof(const uint8_t *);
	hipMalloc(&dev_pmasks_beg, bufsize);
	hipMemcpy(dev_pmasks_beg, &pmasks[0], bufsize, hipMemcpyHostToDevice);

	const uint8_t *poutmask_beg = poutmask + tmp_xbeg;
	int *pdist_beg = pdist + tmp_xbeg;
	const int *pdist_prev_beg = pdist_prev + tmp_xbeg;
	uint8_t *pnums_beg = pnums + tmp_xbeg;
	const uint8_t *pnums_prev_beg = pnums_prev + tmp_xbeg;

	/*hipEvent_t start, kernel;
	hipEventCreate(&start);
	hipEventCreate(&kernel);
	hipEventRecord(start, 0);*/
	kernel_find_seamdistances_cycle_x<<<grid_dim, block_dim>>>(dev_pmasks_beg, numimages, poutmask_beg, pdist_beg, pdist_prev_beg, pnums_beg, pnums_prev_beg, l_straight, l_diag, size);
	/*hipEventRecord(kernel, 0);
	hipEventSynchronize(kernel);

	float time_kernel;
	hipEventElapsedTime(&time_kernel, start, kernel);
	printf("cuda time_kernel: %f ms\n", time_kernel);*/
	hipFree(dev_pmasks_beg);
}

void cuda_find_seamdistances_cycle_y_horiz(
	cv::cuda::GpuMat &dist, cv::cuda::GpuMat &mat, const cv::cuda::GpuMat &outmask, const std::vector<cv::cuda::GpuMat> &masks,
	int shift, int ybeg, int yend, int xbeg, int xend,
	int l_straight)
{
	int size = yend - ybeg;
	if (size < 1)
		return;

	int nthreads = 256;
	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(calc_drid_dim(size, block_dim.x * block_dim.y), 1);

	cv::cuda::PtrStep<int> ptr_dist = dist;
	cv::cuda::PtrStep<uint8_t> ptr_mat = mat;
	const cv::cuda::PtrStep<uint8_t> ptr_outmask = outmask;

	int numimages = masks.size();
	std::vector<cv::cuda::PtrStep<uint8_t> > ptr_masks(numimages);
	for (int i = 0; i < numimages; ++i)
		ptr_masks[i] = masks[i];
	cv::cuda::PtrStep<uint8_t> *dev_ptr_masks;
	int bufsize = numimages * sizeof(cv::cuda::PtrStep<uint8_t>);
	hipMalloc(&dev_ptr_masks, bufsize);
	hipMemcpy(dev_ptr_masks, &ptr_masks[0], bufsize, hipMemcpyHostToDevice);

	/*hipEvent_t start, kernel;
	hipEventCreate(&start);
	hipEventCreate(&kernel);
	hipEventRecord(start, 0);*/
	kernel_find_seamdistances_cycle_y_horiz<<<grid_dim, block_dim>>>(ptr_dist, ptr_mat, ptr_outmask, dev_ptr_masks, shift, ybeg, size, xbeg, xend, l_straight);
	/*hipEventRecord(kernel, 0);
	hipEventSynchronize(kernel);

	float time_kernel;
	hipEventElapsedTime(&time_kernel, start, kernel);
	printf("cuda time_kernel: %f ms\n", time_kernel);*/
	hipFree(dev_ptr_masks);
}

void cuda_extract_masks(std::vector<std::vector<cv::cuda::GpuMat> > &cvmaskpyramids, const cv::cuda::GpuMat &cvseams, int mask_value)
{
	int size = cvseams.cols * cvseams.rows;
	if (size < 1)
		return;

	int nthreads = 256;
	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(calc_drid_dim(size, block_dim.x * block_dim.y), 1);

	cv::cuda::PtrStepSz<uint8_t> ptr_cvseams = cvseams;

	int numimages = cvmaskpyramids.size();
	std::vector<cv::cuda::PtrStepSz<uint8_t> > ptr_masks(numimages);
	for (int i = 0; i < numimages; ++i)
		ptr_masks[i] = cvmaskpyramids[i][0];
	cv::cuda::PtrStepSz<uint8_t> *dev_ptr_masks;
	int bufsize = numimages * sizeof(cv::cuda::PtrStepSz<uint8_t>);
	hipMalloc(&dev_ptr_masks, bufsize);
	hipMemcpy(dev_ptr_masks, &ptr_masks[0], bufsize, hipMemcpyHostToDevice);

	/*hipEvent_t start, kernel;
	hipEventCreate(&start);
	hipEventCreate(&kernel);
	hipEventRecord(start, 0);*/
	kernel_extract_masks<<<grid_dim, block_dim>>>(dev_ptr_masks, ptr_cvseams, mask_value);
	/*hipEventRecord(kernel, 0);
	hipEventSynchronize(kernel);

	float time_kernel;
	hipEventElapsedTime(&time_kernel, start, kernel);
	printf("cuda time_kernel: %f ms\n", time_kernel);*/
	hipFree(dev_ptr_masks);
}

/*
__device__ __constant__ int MAXITER = 100;
__device__ __constant__ real_t M_PI_DEG = 180.0f;
__device__ __constant__ real_t M_EPSILON  = 1e-6f; //enough precosion for --use_fast_math
#undef M_PI
__device__ __constant__ real_t M_PI = 3.14159265358979323846;

__device__ __constant__ int cwhere[4];

static const int nthreads = 256;



__global__ void kernel_init_mat(const diy::Point *map, diy::Point *ocv_map, size_t rows, size_t cols, size_t bufsize, real_t finish_cx, real_t finish_cy, size_t offset)
{
	size_t pt_index = blockIdx.x * blockDim.x + threadIdx.x + offset;

	extern __shared__ diy::Point sh_mem_point[];
	diy::Point *shvar = &sh_mem_point[0];

	while (pt_index < bufsize + offset)
	{
		size_t y = pt_index / cols;
		size_t x = pt_index - y * cols;

		shvar[threadIdx.x] = map[y * cols + x];
		
		shvar[threadIdx.x].x = float(shvar[threadIdx.x].x + finish_cx);
		shvar[threadIdx.x].y = float(shvar[threadIdx.x].y + finish_cy);
		
		ocv_map[pt_index-offset] = shvar[threadIdx.x];
		//ocv_map[pt_index].x = float(invar.x + finish_cx);
		//ocv_map[pt_index].y = float(invar.y + finish_cy);

		pt_index += blockDim.x * gridDim.x;
	}
}

static inline int func_finalize(void *stream)
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("cuda enter status: %s\n", hipGetErrorString(error));
		return -1;
	}
	error = hipStreamSynchronize(*(hipStream_t*)stream);
	if (error != hipSuccess)
	{
		printf("cuda exit status: %s\n", hipGetErrorString(error));
		return -1;
	}
	return 0;
}

void cuda_StreamSynchronize(void *stream)
{
	hipStreamSynchronize(*(hipStream_t*)stream);
}


//API
void cuda_init()
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	printf("Found %d devices\n", deviceCount);
	hipDeviceSynchronize();
}

void* cuda_stream_create()
{
	hipStream_t *stream = (hipStream_t*)malloc(sizeof(hipStream_t));
	if (!stream)
		return 0;
	hipError_t res = hipStreamCreate(stream);
	if (res == hipSuccess)
		return stream;
	else
		return 0;
}

void cuda_stream_destroy(void* stream)
{
	hipError_t res = hipStreamDestroy(*((hipStream_t*)stream));
	free(stream);
	if (res != hipSuccess)
		; //emit error
}

diy::Point* cuda_points_alloc(int size)
{
	diy::Point *ptr = 0;
	hipError_t res = hipMalloc(&ptr, size * sizeof(diy::Point));
	if (res == hipSuccess)
		return ptr;
	else
		return 0;
}

diy::Point* cuda_host_alloc(int size)
{
	diy::Point *ptr = 0;
	hipError_t res = hipHostAlloc((void**)&ptr, size * sizeof(diy::Point), hipHostMallocDefault);
	if (res == hipSuccess)
		return ptr;
	else
		return 0;
}

void cuda_points_free(diy::Point *points)
{
	hipError_t res = hipFree(points);
	if (res != hipSuccess)
		; //emit error
}

void cuda_host_free(diy::Point *points)
{
	hipError_t res = hipHostFree(points);
	if (res != hipSuccess)
		; //emit error
}

void cuda_points_to_device(const diy::Point *host_data, diy::Point *device_data, int size, void *stream)
{
	//TODO: error checking
	hipMemcpyAsync(device_data, host_data, size * sizeof(diy::Point), hipMemcpyHostToDevice, *((hipStream_t*)stream));
}

void cuda_points_from_device(const diy::Point *device_data, diy::Point *host_data, int size, void *stream)
{
	//TODO: error checking
	hipMemcpyAsync(host_data, device_data, size * sizeof(diy::Point), hipMemcpyDeviceToHost, *((hipStream_t*)stream));
}








int cuda_init_mat(
	const diy::Point *map, diy::Point *ocv_map,
	size_t rows, size_t cols, int channels, real_t finish_cx, real_t finish_cy)
{
	
	hipEvent_t start, props, maloc, kernel, fre;
hipEventCreate(&start);
hipEventCreate(&props);
hipEventCreate(&maloc);
hipEventCreate(&kernel);
hipEventCreate(&fre);

hipEventRecord(start, 0);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	int nthreads = 256;
	int nblocks = prop.multiProcessorCount * 128;

	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(nblocks, 1);

	int shared_memory = nthreads * sizeof(diy::Point);
	int size = rows * cols;
	diy::Point *map_dev;
	diy::Point *ocv_map_dev;

hipEventRecord(props, 0);
hipEventSynchronize(props);
	size_t N = 2;
	size_t blocksize = (size / N) + 1;
	
	hipMalloc(&map_dev, blocksize * sizeof(diy::Point));
	hipMalloc(&ocv_map_dev, blocksize * sizeof(diy::Point));

hipEventRecord(maloc, 0);
hipEventSynchronize(maloc);

	printf("size = %d\n", size);
	for (size_t block = 0; block < N; ++block)
	{
		size_t bufsize = (block == N - 1) ? (size - block*blocksize) : blocksize;
		size_t offset = block * blocksize;
		printf("bufsize[%d] = %d\n", block, bufsize);
		printf("offset[%d] = %d\n", block, offset);

		hipMemcpy(map_dev, &map[offset], bufsize * sizeof(diy::Point), hipMemcpyHostToDevice);

		kernel_init_mat<<<nblocks, block_dim, shared_memory>>>(map_dev, ocv_map_dev, rows, cols, bufsize, finish_cx, finish_cy, offset);

		hipMemcpy(&ocv_map[offset], ocv_map_dev, bufsize * sizeof(diy::Point), hipMemcpyDeviceToHost);
	
		printf("ok\n");
	}


hipEventRecord(kernel, 0);
hipEventSynchronize(kernel);

	hipFree(map_dev);
	hipFree(ocv_map_dev);

hipEventRecord(fre, 0);
hipEventSynchronize(fre);

float time_props, time_maloc, time_kernel, time_fre, time_all;
hipEventElapsedTime(&time_props, start, props);
hipEventElapsedTime(&time_maloc, props, maloc);
hipEventElapsedTime(&time_kernel, maloc, kernel);
hipEventElapsedTime(&time_fre, kernel, fre);
hipEventElapsedTime(&time_all, start, fre);

printf("cuda time_props: %f ms\n", time_props);
printf("cuda time_maloc: %f ms\n", time_maloc);
printf("cuda time_kernel: %f ms\n", time_kernel);
printf("cuda time_fre: %f ms\n", time_fre);
printf("cuda time_all(%f): %f ms\n\n", (rows*cols*sizeof(diy::Point)) / (1024.0*1024.0), time_all);

	return 0;
}
*/