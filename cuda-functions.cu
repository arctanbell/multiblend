#include "hip/hip_runtime.h"
#include "cuda-functions.h"
#ifndef NO_CUDA
#include <hip/hip_runtime.h>

/*
__device__ __constant__ int MAXITER = 100;
__device__ __constant__ real_t M_PI_DEG = 180.0f;
__device__ __constant__ real_t M_EPSILON  = 1e-6f; //enough precosion for --use_fast_math
#undef M_PI
__device__ __constant__ real_t M_PI = 3.14159265358979323846;

__device__ __constant__ int cwhere[4];

static const int nthreads = 256;



__global__ void kernel_init_mat(const diy::Point *map, diy::Point *ocv_map, size_t rows, size_t cols, size_t bufsize, real_t finish_cx, real_t finish_cy, size_t offset)
{
	size_t pt_index = blockIdx.x * blockDim.x + threadIdx.x + offset;

	extern __shared__ diy::Point sh_mem_point[];
	diy::Point *shvar = &sh_mem_point[0];

	while (pt_index < bufsize + offset)
	{
		size_t y = pt_index / cols;
		size_t x = pt_index - y * cols;

		shvar[threadIdx.x] = map[y * cols + x];
		
		shvar[threadIdx.x].x = float(shvar[threadIdx.x].x + finish_cx);
		shvar[threadIdx.x].y = float(shvar[threadIdx.x].y + finish_cy);
		
		ocv_map[pt_index-offset] = shvar[threadIdx.x];
		//ocv_map[pt_index].x = float(invar.x + finish_cx);
		//ocv_map[pt_index].y = float(invar.y + finish_cy);

		pt_index += blockDim.x * gridDim.x;
	}
}



















static inline int func_finalize(void *stream)
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("cuda enter status: %s\n", hipGetErrorString(error));
		return -1;
	}
	error = hipStreamSynchronize(*(hipStream_t*)stream);
	if (error != hipSuccess)
	{
		printf("cuda exit status: %s\n", hipGetErrorString(error));
		return -1;
	}
	return 0;
}

void cuda_StreamSynchronize(void *stream)
{
	hipStreamSynchronize(*(hipStream_t*)stream);
}


//API
void cuda_init()
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	printf("Found %d devices\n", deviceCount);
	hipDeviceSynchronize();
}

void* cuda_stream_create()
{
	hipStream_t *stream = (hipStream_t*)malloc(sizeof(hipStream_t));
	if (!stream)
		return 0;
	hipError_t res = hipStreamCreate(stream);
	if (res == hipSuccess)
		return stream;
	else
		return 0;
}

void cuda_stream_destroy(void* stream)
{
	hipError_t res = hipStreamDestroy(*((hipStream_t*)stream));
	free(stream);
	if (res != hipSuccess)
		; //emit error
}

diy::Point* cuda_points_alloc(int size)
{
	diy::Point *ptr = 0;
	hipError_t res = hipMalloc(&ptr, size * sizeof(diy::Point));
	if (res == hipSuccess)
		return ptr;
	else
		return 0;
}

diy::Point* cuda_host_alloc(int size)
{
	diy::Point *ptr = 0;
	hipError_t res = hipHostAlloc((void**)&ptr, size * sizeof(diy::Point), hipHostMallocDefault);
	if (res == hipSuccess)
		return ptr;
	else
		return 0;
}

void cuda_points_free(diy::Point *points)
{
	hipError_t res = hipFree(points);
	if (res != hipSuccess)
		; //emit error
}

void cuda_host_free(diy::Point *points)
{
	hipError_t res = hipHostFree(points);
	if (res != hipSuccess)
		; //emit error
}

void cuda_points_to_device(const diy::Point *host_data, diy::Point *device_data, int size, void *stream)
{
	//TODO: error checking
	hipMemcpyAsync(device_data, host_data, size * sizeof(diy::Point), hipMemcpyHostToDevice, *((hipStream_t*)stream));
}

void cuda_points_from_device(const diy::Point *device_data, diy::Point *host_data, int size, void *stream)
{
	//TODO: error checking
	hipMemcpyAsync(host_data, device_data, size * sizeof(diy::Point), hipMemcpyDeviceToHost, *((hipStream_t*)stream));
}








int cuda_init_mat(
	const diy::Point *map, diy::Point *ocv_map,
	size_t rows, size_t cols, int channels, real_t finish_cx, real_t finish_cy)
{
	
	hipEvent_t start, props, maloc, kernel, fre;
hipEventCreate(&start);
hipEventCreate(&props);
hipEventCreate(&maloc);
hipEventCreate(&kernel);
hipEventCreate(&fre);

hipEventRecord(start, 0);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	int nthreads = 256;
	int nblocks = prop.multiProcessorCount * 128;

	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(nblocks, 1);

	int shared_memory = nthreads * sizeof(diy::Point);
	int size = rows * cols;
	diy::Point *map_dev;
	diy::Point *ocv_map_dev;

hipEventRecord(props, 0);
hipEventSynchronize(props);
	size_t N = 2;
	size_t blocksize = (size / N) + 1;
	
	hipMalloc(&map_dev, blocksize * sizeof(diy::Point));
	hipMalloc(&ocv_map_dev, blocksize * sizeof(diy::Point));

hipEventRecord(maloc, 0);
hipEventSynchronize(maloc);

	printf("size = %d\n", size);
	for (size_t block = 0; block < N; ++block)
	{
		size_t bufsize = (block == N - 1) ? (size - block*blocksize) : blocksize;
		size_t offset = block * blocksize;
		printf("bufsize[%d] = %d\n", block, bufsize);
		printf("offset[%d] = %d\n", block, offset);

		hipMemcpy(map_dev, &map[offset], bufsize * sizeof(diy::Point), hipMemcpyHostToDevice);

		kernel_init_mat<<<nblocks, block_dim, shared_memory>>>(map_dev, ocv_map_dev, rows, cols, bufsize, finish_cx, finish_cy, offset);

		hipMemcpy(&ocv_map[offset], ocv_map_dev, bufsize * sizeof(diy::Point), hipMemcpyDeviceToHost);
	
		printf("ok\n");
	}


hipEventRecord(kernel, 0);
hipEventSynchronize(kernel);

	hipFree(map_dev);
	hipFree(ocv_map_dev);

hipEventRecord(fre, 0);
hipEventSynchronize(fre);

float time_props, time_maloc, time_kernel, time_fre, time_all;
hipEventElapsedTime(&time_props, start, props);
hipEventElapsedTime(&time_maloc, props, maloc);
hipEventElapsedTime(&time_kernel, maloc, kernel);
hipEventElapsedTime(&time_fre, kernel, fre);
hipEventElapsedTime(&time_all, start, fre);

printf("cuda time_props: %f ms\n", time_props);
printf("cuda time_maloc: %f ms\n", time_maloc);
printf("cuda time_kernel: %f ms\n", time_kernel);
printf("cuda time_fre: %f ms\n", time_fre);
printf("cuda time_all(%f): %f ms\n\n", (rows*cols*sizeof(diy::Point)) / (1024.0*1024.0), time_all);

	return 0;
}
*/
#endif
