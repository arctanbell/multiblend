#include "hip/hip_runtime.h"
#include "cuda-functions.h"
#ifndef NO_CUDA
#include <hip/hip_runtime.h>
#include <opencv2/cudaarithm.hpp>

#include <cstdio>

//helpers
static int calc_drid_dim(int array_size, int block_size)
{
	return (array_size - 1) / block_size + 1;
}

void cuda_get_memory(size_t *free, size_t *total)
{
	hipMemGetInfo(free, total);
}

__global__ void kernel_find_distances_cycle_y_horiz(cv::cuda::PtrStep<float> ptr_dist, cv::cuda::PtrStep<uint8_t> ptr_mat, cv::cuda::PtrStep<uint8_t> ptr_mask, int shift, int ybeg, int size, int xbeg, int xend, int l_straight)
{
	size_t y = blockIdx.x * blockDim.x + threadIdx.x + ybeg;
	
	if (y >= ybeg + size)
		return;

	auto pdist = ptr_dist.ptr(y);
	auto pmat = ptr_mat.ptr(y);
	auto pmask = ptr_mask.ptr(y);

	int x = xbeg;
	while (x != xend)
	{
		if (pmask[x])
		{
			x += shift;
			continue;
		}
		if (pdist[x - shift] + l_straight < pdist[x])
		{
			pdist[x] = pdist[x - shift] + l_straight;
			pmat[x] = pmat[x - shift];
		}
		x += shift;
	}
}

__global__ void kernel_find_distances_cycle_x(float *pdist, uint8_t *pnums, const uint8_t *pmask, const float *pdist_prev, const uint8_t *pnums_prev, int l_straight, int l_diag, int size)
{
	size_t x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x >= size || pmask[x] || pdist[x] == 0)
		return;

	if (pdist_prev[x] + l_straight < pdist[x])
	{
		pdist[x] = pdist_prev[x] + l_straight;
		pnums[x] = pnums_prev[x];
	}

	if (x != 0)
	{
		if (pdist_prev[x - 1] + l_diag < pdist[x])
		{
			pdist[x] = pdist_prev[x - 1] + l_diag;
			pnums[x] = pnums_prev[x - 1];
		}
	}

	if (x != (size - 1))
	{
		if (pdist_prev[x + 1] + l_diag < pdist[x])
		{
			pdist[x] = pdist_prev[x + 1] + l_diag;
			pnums[x] = pnums_prev[x + 1];
		}
	}
}

__global__ void kernel_init_seamdist(cv::cuda::PtrStepSz<int> ptr_dist, cv::cuda::PtrStep<uint8_t> ptr_nums, cv::cuda::PtrStep<uint8_t> ptr_outmask, cv::cuda::PtrStep<uint8_t> *ptr_masks, int size_masks)
{
	size_t pt = blockIdx.x * blockDim.x + threadIdx.x;
	if (pt >= ptr_dist.cols * ptr_dist.rows)
		return;
	int y = pt / ptr_dist.cols;
	int x = pt - ptr_dist.cols * y;

	int dist_max = 2 * (ptr_dist.cols + ptr_dist.rows);

	int* pdist = ptr_dist.ptr(y);
	uint8_t* pnums = ptr_nums.ptr(y);
	uint8_t* poutmask = ptr_outmask.ptr(y);

	int count = 0;
	int num = 0;
	for (int i = 0; i < size_masks; ++i)
		if (ptr_masks[i].ptr(y)[x]) //1-visible
		{
			++count;
			num = i;
		}

	if (count == 0)
		poutmask[x] = 1;
	else
		poutmask[x] = 0;

	if (count == 1)
	{
		pdist[x] = 0;
		pnums[x] = num;
	}
	else
	{
		pdist[x] = dist_max;
		pnums[x] = 0;
	}
}

__global__ void kernel_find_seamdistances_cycle_x(const uint8_t **pmasks, int numimages, const uint8_t *poutmask, int *pdist, const int *pdist_prev, uint8_t *pnums, const uint8_t *pnums_prev, int l_straight, int l_diag, int size)
{
	size_t x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x >= size || pdist[x] == 0)
		return;

	if (pdist_prev[x] + l_straight < pdist[x] && (pmasks[pnums_prev[x]][x] || poutmask[x]))
	{
		pdist[x] = pdist_prev[x] + l_straight;
		pnums[x] = pnums_prev[x];
	}

	if (x != 0)
	{
		if (pdist_prev[x - 1] + l_diag < pdist[x] && (pmasks[pnums_prev[x - 1]][x] || poutmask[x]))
		{
			pdist[x] = pdist_prev[x - 1] + l_diag;
			pnums[x] = pnums_prev[x - 1];
		}
	}

	if (x != (size - 1))
	{
		if (pdist_prev[x + 1] + l_diag < pdist[x] && (pmasks[pnums_prev[x + 1]][x] || poutmask[x]))
		{
			pdist[x] = pdist_prev[x + 1] + l_diag;
			pnums[x] = pnums_prev[x + 1];
		}
	}

}

__global__ void kernel_find_seamdistances_cycle_y_horiz(cv::cuda::PtrStep<int> ptr_dist, cv::cuda::PtrStep<uint8_t> ptr_mat, cv::cuda::PtrStep<uint8_t> ptr_outmask, cv::cuda::PtrStep<uint8_t> *ptr_masks, int shift, int ybeg, int size, int xbeg, int xend, int l_straight)
{
	size_t y = blockIdx.x * blockDim.x + threadIdx.x + ybeg;
	if (y >= ybeg + size)
		return;

	auto pdist = ptr_dist.ptr(y);
	auto pmat = ptr_mat.ptr(y);
	auto poutmask = ptr_outmask.ptr(y);
		
	int x = xbeg;
	while (x != xend)
	{
		if (pdist[x] == 0)
		{
			x += shift;
			continue;
		}
		if (pdist[x - shift] + l_straight < pdist[x] && (ptr_masks[pmat[x - shift]].ptr(y)[x] || poutmask[x]))
		{
			pdist[x] = pdist[x - shift] + l_straight;
			pmat[x] = pmat[x - shift];
		}
		x += shift;
	}
}

__global__ void kernel_extract_masks(cv::cuda::PtrStepSz<uint8_t> *ptr_masks, cv::cuda::PtrStepSz<uint8_t> ptr_cvseams, int mask_value)
{
	size_t pt = blockIdx.x * blockDim.x + threadIdx.x;
	if (pt >= ptr_cvseams.cols * ptr_cvseams.rows)
		return;
	int y = pt / ptr_cvseams.cols;
	int x = pt - ptr_cvseams.cols * y;

	auto pseam = ptr_cvseams.ptr(y);
	ptr_masks[pseam[x]].ptr(y)[x] = mask_value;
}

__global__ void kernel_mask_into_output(cv::cuda::PtrStepSz<short> ptr_out, cv::cuda::PtrStepSz<short> ptr_in, cv::cuda::PtrStepSz<uint8_t> ptr_mask, cv::Size ofs, int max_value)
{
	size_t pt = blockIdx.x * blockDim.x + threadIdx.x;
	if (pt >= ptr_in.cols * ptr_in.rows)
		return;
	int y = pt / ptr_in.cols;
	int x = pt - ptr_in.cols * y;

	auto pin = ptr_in.ptr(y);
	auto pout = ofs.width + ptr_out.ptr(y + ofs.height);
	auto pmask = ofs.width + ptr_mask.ptr(y + ofs.height);

	pout[x] += (int)((float)pmask[x] * pin[x] / max_value + 0.5);
}

__global__ void kernel_pyrDown_x(cv::cuda::PtrStepSz<short> ptr_umat, cv::cuda::PtrStepSz<short> ptr_tmp)
{
	size_t pt = blockIdx.x * blockDim.x + threadIdx.x;
	if (pt >= ptr_tmp.cols * ptr_tmp.rows)
		return;
	int y = pt / ptr_tmp.cols;
	int x = pt - ptr_tmp.cols * y;

	auto ptmp = ptr_tmp.ptr(y);
	auto pumat = ptr_umat.ptr(y);

	int temp = 2 * pumat[2 * x];

	if (x != 0)
		temp += pumat[2 * x - 1];
	else
		temp += pumat[2 * x];

	if (x != ptr_tmp.cols - 1)
		temp += pumat[2 * x + 1];
	else
		temp += pumat[2 * x];

	ptmp[x] = temp;
}

__global__ void kernel_pyrDown_y(cv::cuda::PtrStepSz<short> ptr_tmp, cv::cuda::PtrStepSz<short> ptr_lmat)
{
	size_t pt = blockIdx.x * blockDim.x + threadIdx.x;
	if (pt >= ptr_lmat.cols * ptr_lmat.rows)
		return;
	
	int y = pt / ptr_lmat.cols;
	int x = pt - ptr_lmat.cols * y;
	
	int temp = 2 * ptr_tmp.ptr(2 * y)[x];
	
	if (y != 0)
		temp += ptr_tmp.ptr(2 * y - 1)[x];
	else
		temp += ptr_tmp.ptr(2 * y)[x];
	
	if (y != ptr_lmat.rows - 1)
		temp += ptr_tmp.ptr(2 * y + 1)[x];
	else
		temp += ptr_tmp.ptr(2 * y)[x];

	temp = (temp + 8) / 16;

	ptr_lmat.ptr(y)[x] = temp;
}

__global__ void kernel_pyrUp_x(cv::cuda::PtrStepSz<short> ptr_lmat, cv::cuda::PtrStepSz<short> ptr_tmp)
{
	size_t pt = blockIdx.x * blockDim.x + threadIdx.x;
	if (pt >= ptr_lmat.cols * ptr_lmat.rows)
		return;

	int y = pt / ptr_lmat.cols;
	int x = pt - ptr_lmat.cols * y;

	auto plmat = ptr_lmat.ptr(y);
	auto ptmp = ptr_tmp.ptr(y);
	
	int temp = plmat[x];
	ptmp[2 * x] = temp;

	if (x != ptr_lmat.cols - 1)
		ptmp[2 * x + 1] = (temp + plmat[x + 1] + 1) / 2;
}

__global__ void kernel_pyrUp_y(cv::cuda::PtrStepSz<short> ptr_tmp, cv::cuda::PtrStepSz<short> ptr_umat)
{
	size_t pt = blockIdx.x * blockDim.x + threadIdx.x;
	if (pt >= ptr_tmp.cols * ptr_tmp.rows)
		return;

	int y = pt / ptr_tmp.cols;
	int x = pt - ptr_tmp.cols * y;

	auto pumat = ptr_umat.ptr(y);

	int temp = ptr_tmp.ptr(y)[x];
	ptr_umat.ptr(2 * y)[x] = temp;
	
	if (y != ptr_tmp.rows - 1)
		ptr_umat.ptr(2 * y + 1)[x] = (temp + ptr_tmp.ptr(y + 1)[x] + 1) / 2;
}

__global__ void kernel_dither(cv::cuda::PtrStepSz<short> ptr_in, cv::cuda::PtrStepSz<uint8_t> ptr_out, int denom)
{
	size_t pt = blockIdx.x * blockDim.x + threadIdx.x;
	if (pt >= ptr_in.cols * ptr_in.rows)
		return;
	int y = pt / ptr_in.cols;
	int x = pt - ptr_in.cols * y;

	int temp = ptr_in.ptr(y)[x];
	//temp += random_number;
	temp /= denom;
	if (temp < 0) temp = 0;
	else if (temp > 255) temp = 255;
	ptr_out.ptr(y)[x] = temp;
}

void cuda_find_distances_cycle_y_horiz(
	cv::cuda::GpuMat &dist, cv::cuda::GpuMat &mat, const cv::cuda::GpuMat &mask,
	int shift, int ybeg, int yend, int xbeg, int xend,
	int l_straight)
{
	int size = yend - ybeg;
	if (size < 1)
		return;

	int nthreads = 256;
	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(calc_drid_dim(size, block_dim.x * block_dim.y), 1);

	cv::cuda::PtrStep<float> ptr_dist = dist;
	cv::cuda::PtrStep<uint8_t> ptr_mat = mat;
	cv::cuda::PtrStep<uint8_t> ptr_mask = mask;

	/*hipEvent_t start, kernel;
	hipEventCreate(&start);
	hipEventCreate(&kernel);
	hipEventRecord(start, 0);*/
	kernel_find_distances_cycle_y_horiz <<<grid_dim, block_dim >>>(ptr_dist, ptr_mat, ptr_mask, shift, ybeg, size, xbeg, xend, l_straight);
	/*hipEventRecord(kernel, 0);
	hipEventSynchronize(kernel);

	float time_kernel;
	hipEventElapsedTime(&time_kernel, start, kernel);
	printf("cuda time_kernel: %f ms\n", time_kernel);*/
}

void cuda_find_distances_cycle_x(
	const uint8_t *pmask, float *pdist, const float *pdist_prev, uint8_t *pnums, const uint8_t *pnums_prev,
	int tmp_xbeg, int tmp_xend,
	int l_straight, int l_diag)
{
	int size = tmp_xend - tmp_xbeg;
	if (size < 1)
		return;

	int nthreads = 256;
	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(calc_drid_dim(size, block_dim.x * block_dim.y), 1);
	
	float *pdist_beg = pdist + tmp_xbeg;
	uint8_t *pnums_beg = pnums + tmp_xbeg;
	const uint8_t *pmask_beg = pmask + tmp_xbeg;
	const float *pdist_prev_beg = pdist_prev + tmp_xbeg;
	const uint8_t *pnums_prev_beg = pnums_prev + tmp_xbeg;

	/*hipEvent_t start, kernel;
	hipEventCreate(&start);
	hipEventCreate(&kernel);
	hipEventRecord(start, 0);*/
	kernel_find_distances_cycle_x<<<grid_dim, block_dim>>>(pdist_beg, pnums_beg, pmask_beg, pdist_prev_beg, pnums_prev_beg, l_straight, l_diag, size);
	/*hipEventRecord(kernel, 0);
	hipEventSynchronize(kernel);

	float time_kernel;
	hipEventElapsedTime(&time_kernel, start, kernel);
	printf("cuda time_kernel: %f ms\n", time_kernel);*/
}

void cuda_init_seamdist(cv::cuda::GpuMat &dist, cv::cuda::GpuMat &nums, cv::cuda::GpuMat &outmask, const std::vector<cv::cuda::GpuMat> &masks)
{
	int size = dist.cols * dist.rows;
	if (size < 1)
		return;

	int nthreads = 256;
	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(calc_drid_dim(size, block_dim.x * block_dim.y), 1);

	cv::cuda::PtrStepSz<int> ptr_dist = dist;
	cv::cuda::PtrStep<uint8_t> ptr_nums = nums;
	cv::cuda::PtrStep<uint8_t> ptr_outmask = outmask;
	int numimages = masks.size();
	std::vector<cv::cuda::PtrStep<uint8_t> > ptr_masks(numimages);
	for (int i = 0; i < numimages; ++i)
		ptr_masks[i] = masks[i];

	cv::cuda::PtrStep<uint8_t> *dev_ptr_masks;
	int bufsize = numimages * sizeof(cv::cuda::PtrStep<uint8_t>);
	hipMalloc(&dev_ptr_masks, bufsize);
	hipMemcpy(dev_ptr_masks, &ptr_masks[0], bufsize, hipMemcpyHostToDevice);

	/*hipEvent_t start, kernel;
	hipEventCreate(&start);
	hipEventCreate(&kernel);
	hipEventRecord(start, 0);*/
	kernel_init_seamdist<<<grid_dim, block_dim>>>(ptr_dist, ptr_nums, ptr_outmask, dev_ptr_masks, numimages);
	/*hipEventRecord(kernel, 0);
	hipEventSynchronize(kernel);

	float time_kernel;
	hipEventElapsedTime(&time_kernel, start, kernel);
	printf("cuda time_kernel: %f ms\n", time_kernel);*/
	hipFree(dev_ptr_masks);
}

void cuda_find_seamdistances_cycle_x(
	const std::vector<const uint8_t*> &pmasks, const uint8_t *poutmask, int *pdist, const int *pdist_prev, uint8_t *pnums, const uint8_t *pnums_prev,
	int tmp_xbeg, int tmp_xend,
	int l_straight, int l_diag)
{
	int size = tmp_xend - tmp_xbeg;
	if (size < 1)
		return;

	int nthreads = 256;
	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(calc_drid_dim(size, block_dim.x * block_dim.y), 1);

	int numimages = pmasks.size();
	std::vector<const uint8_t *> pmasks_beg(numimages);
	for (int i = 0; i < numimages; ++i)
		pmasks_beg[i] = pmasks[i] + tmp_xbeg;
	const uint8_t **dev_pmasks_beg;
	int bufsize = numimages * sizeof(const uint8_t *);
	hipMalloc(&dev_pmasks_beg, bufsize);
	hipMemcpy(dev_pmasks_beg, &pmasks[0], bufsize, hipMemcpyHostToDevice);

	const uint8_t *poutmask_beg = poutmask + tmp_xbeg;
	int *pdist_beg = pdist + tmp_xbeg;
	const int *pdist_prev_beg = pdist_prev + tmp_xbeg;
	uint8_t *pnums_beg = pnums + tmp_xbeg;
	const uint8_t *pnums_prev_beg = pnums_prev + tmp_xbeg;

	/*hipEvent_t start, kernel;
	hipEventCreate(&start);
	hipEventCreate(&kernel);
	hipEventRecord(start, 0);*/
	kernel_find_seamdistances_cycle_x<<<grid_dim, block_dim>>>(dev_pmasks_beg, numimages, poutmask_beg, pdist_beg, pdist_prev_beg, pnums_beg, pnums_prev_beg, l_straight, l_diag, size);
	/*hipEventRecord(kernel, 0);
	hipEventSynchronize(kernel);

	float time_kernel;
	hipEventElapsedTime(&time_kernel, start, kernel);
	printf("cuda time_kernel: %f ms\n", time_kernel);*/
	hipFree(dev_pmasks_beg);
}

void cuda_find_seamdistances_cycle_y_horiz(
	cv::cuda::GpuMat &dist, cv::cuda::GpuMat &mat, const cv::cuda::GpuMat &outmask, const std::vector<cv::cuda::GpuMat> &masks,
	int shift, int ybeg, int yend, int xbeg, int xend,
	int l_straight)
{
	int size = yend - ybeg;
	if (size < 1)
		return;

	int nthreads = 256;
	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(calc_drid_dim(size, block_dim.x * block_dim.y), 1);

	cv::cuda::PtrStep<int> ptr_dist = dist;
	cv::cuda::PtrStep<uint8_t> ptr_mat = mat;
	const cv::cuda::PtrStep<uint8_t> ptr_outmask = outmask;

	int numimages = masks.size();
	std::vector<cv::cuda::PtrStep<uint8_t> > ptr_masks(numimages);
	for (int i = 0; i < numimages; ++i)
		ptr_masks[i] = masks[i];
	cv::cuda::PtrStep<uint8_t> *dev_ptr_masks;
	int bufsize = numimages * sizeof(cv::cuda::PtrStep<uint8_t>);
	hipMalloc(&dev_ptr_masks, bufsize);
	hipMemcpy(dev_ptr_masks, &ptr_masks[0], bufsize, hipMemcpyHostToDevice);

	/*hipEvent_t start, kernel;
	hipEventCreate(&start);
	hipEventCreate(&kernel);
	hipEventRecord(start, 0);*/
	kernel_find_seamdistances_cycle_y_horiz<<<grid_dim, block_dim>>>(ptr_dist, ptr_mat, ptr_outmask, dev_ptr_masks, shift, ybeg, size, xbeg, xend, l_straight);
	/*hipEventRecord(kernel, 0);
	hipEventSynchronize(kernel);

	float time_kernel;
	hipEventElapsedTime(&time_kernel, start, kernel);
	printf("cuda time_kernel: %f ms\n", time_kernel);*/
	hipFree(dev_ptr_masks);
}



void cuda_extract_masks(std::vector<std::vector<cv::cuda::GpuMat> > &cvmaskpyramids, const cv::cuda::GpuMat &cvseams, int mask_value)
{
	int size = cvseams.cols * cvseams.rows;
	if (size < 1)
		return;

	int nthreads = 256;
	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(calc_drid_dim(size, block_dim.x * block_dim.y), 1);

	cv::cuda::PtrStepSz<uint8_t> ptr_cvseams = cvseams;

	int numimages = cvmaskpyramids.size();
	std::vector<cv::cuda::PtrStepSz<uint8_t> > ptr_masks(numimages);
	for (int i = 0; i < numimages; ++i)
		ptr_masks[i] = cvmaskpyramids[i][0];
	cv::cuda::PtrStepSz<uint8_t> *dev_ptr_masks;
	int bufsize = numimages * sizeof(cv::cuda::PtrStepSz<uint8_t>);
	hipMalloc(&dev_ptr_masks, bufsize);
	hipMemcpy(dev_ptr_masks, &ptr_masks[0], bufsize, hipMemcpyHostToDevice);

	/*hipEvent_t start, kernel;
	hipEventCreate(&start);
	hipEventCreate(&kernel);
	hipEventRecord(start, 0);*/
	kernel_extract_masks<<<grid_dim, block_dim>>>(dev_ptr_masks, ptr_cvseams, mask_value);
	/*hipEventRecord(kernel, 0);
	hipEventSynchronize(kernel);

	float time_kernel;
	hipEventElapsedTime(&time_kernel, start, kernel);
	printf("cuda time_kernel: %f ms\n", time_kernel);*/
	hipFree(dev_ptr_masks);
}

void cuda_mask_into_output(std::vector<cv::cuda::GpuMat> &outs, const std::vector<cv::cuda::GpuMat> &ins, const cv::cuda::GpuMat &mask, const cv::Size &ofs, int max_value)
{
	int size = ins[0].cols * ins[0].rows;
	if (size < 1)
		return;

	int nthreads = 256;
	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(calc_drid_dim(size, block_dim.x * block_dim.y), 1);

	cv::cuda::PtrStepSz<uint8_t> ptr_mask = mask;

	for (int c = 0; c < 3; ++c)
	{
		cv::cuda::PtrStepSz<short> ptr_in = ins[c];
		cv::cuda::PtrStepSz<short> ptr_out = outs[c];

		/*hipEvent_t start, kernel;
		hipEventCreate(&start);
		hipEventCreate(&kernel);
		hipEventRecord(start, 0);*/
		kernel_mask_into_output<<<grid_dim, block_dim>>>(ptr_out, ptr_in, ptr_mask, ofs, max_value);
		/*hipEventRecord(kernel, 0);
		hipEventSynchronize(kernel);

		float time_kernel;
		hipEventElapsedTime(&time_kernel, start, kernel);
		printf("cuda time_kernel: %f ms\n", time_kernel);*/
	}
}

void cuda_pyrDown(const cv::cuda::GpuMat &umat, cv::cuda::GpuMat &lmat)
{
	cv::cuda::GpuMat tmp(umat.rows, (umat.cols + 1) >> 1, umat.type());
	int sizex = tmp.cols * tmp.rows;
	if (sizex < 1)
		return;

	int nthreads = 256;
	int grid_dim = calc_drid_dim(sizex, nthreads);

	cv::cuda::PtrStepSz<short> ptr_umat = umat;
	cv::cuda::PtrStepSz<short> ptr_tmp = tmp;

	kernel_pyrDown_x<<<grid_dim, nthreads>>>(ptr_umat, ptr_tmp);

	lmat = cv::cuda::GpuMat((umat.rows + 1) >> 1, (umat.cols + 1) >> 1, umat.type());
	cv::cuda::PtrStepSz<short> ptr_lmat = lmat;

	int sizey = lmat.cols * lmat.rows;
	if (sizey < 1)
		return;
	grid_dim = calc_drid_dim(sizey, nthreads);

	kernel_pyrDown_y<<<grid_dim, nthreads>>>(ptr_tmp, ptr_lmat);
}

void cuda_pyrUp(const cv::cuda::GpuMat &lmat, cv::cuda::GpuMat &umat)
{
	int sizex = lmat.cols * lmat.rows;
	if (sizex < 1)
		return;
	int nthreads = 256;
	int grid_dim = calc_drid_dim(sizex, nthreads);

	cv::cuda::GpuMat tmp(lmat.rows, (lmat.cols << 1) - 1, lmat.type());

	cv::cuda::PtrStepSz<short> ptr_lmat = lmat;
	cv::cuda::PtrStepSz<short> ptr_tmp = tmp;

	kernel_pyrUp_x<<<grid_dim, nthreads>>>(ptr_lmat, ptr_tmp);
	
	int sizey = tmp.cols * tmp.rows;
	if (sizey < 1)
		return;

	grid_dim = calc_drid_dim(sizey, nthreads);

	umat = cv::cuda::GpuMat((lmat.rows << 1) - 1, (lmat.cols << 1) - 1, lmat.type());
	cv::cuda::PtrStepSz<short> ptr_umat = umat;

	kernel_pyrUp_y<<<grid_dim, nthreads>>>(ptr_tmp, ptr_umat);
}

void cuda_dither(const cv::cuda::GpuMat &in, cv::cuda::GpuMat &out, int denom)
{
	int size = in.cols * in.rows;
	if (size < 1)
		return;

	int nthreads = 256;
	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(calc_drid_dim(size, block_dim.x * block_dim.y), 1);
	out = cv::cuda::GpuMat(in.rows, in.cols, CV_8U);
	cv::cuda::PtrStepSz<short> ptr_in = in;
	cv::cuda::PtrStepSz<uint8_t> ptr_out = out;

	/*hipEvent_t start, kernel;
	hipEventCreate(&start);
	hipEventCreate(&kernel);
	hipEventRecord(start, 0);*/
	kernel_dither<<<grid_dim, block_dim>>>(ptr_in, ptr_out, denom);
	/*hipEventRecord(kernel, 0);
	hipEventSynchronize(kernel);

	float time_kernel;
	hipEventElapsedTime(&time_kernel, start, kernel);
	printf("cuda time_kernel: %f ms\n", time_kernel);*/
}

/*
__device__ __constant__ int MAXITER = 100;
__device__ __constant__ real_t M_PI_DEG = 180.0f;
__device__ __constant__ real_t M_EPSILON  = 1e-6f; //enough precosion for --use_fast_math
#undef M_PI
__device__ __constant__ real_t M_PI = 3.14159265358979323846;

__device__ __constant__ int cwhere[4];

static const int nthreads = 256;



__global__ void kernel_init_mat(const diy::Point *map, diy::Point *ocv_map, size_t rows, size_t cols, size_t bufsize, real_t finish_cx, real_t finish_cy, size_t offset)
{
	size_t pt_index = blockIdx.x * blockDim.x + threadIdx.x + offset;

	extern __shared__ diy::Point sh_mem_point[];
	diy::Point *shvar = &sh_mem_point[0];

	while (pt_index < bufsize + offset)
	{
		size_t y = pt_index / cols;
		size_t x = pt_index - y * cols;

		shvar[threadIdx.x] = map[y * cols + x];
		
		shvar[threadIdx.x].x = float(shvar[threadIdx.x].x + finish_cx);
		shvar[threadIdx.x].y = float(shvar[threadIdx.x].y + finish_cy);
		
		ocv_map[pt_index-offset] = shvar[threadIdx.x];
		//ocv_map[pt_index].x = float(invar.x + finish_cx);
		//ocv_map[pt_index].y = float(invar.y + finish_cy);

		pt_index += blockDim.x * gridDim.x;
	}
}

static inline int func_finalize(void *stream)
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("cuda enter status: %s\n", hipGetErrorString(error));
		return -1;
	}
	error = hipStreamSynchronize(*(hipStream_t*)stream);
	if (error != hipSuccess)
	{
		printf("cuda exit status: %s\n", hipGetErrorString(error));
		return -1;
	}
	return 0;
}

void cuda_StreamSynchronize(void *stream)
{
	hipStreamSynchronize(*(hipStream_t*)stream);
}


//API
void cuda_init()
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	printf("Found %d devices\n", deviceCount);
	hipDeviceSynchronize();
}

void* cuda_stream_create()
{
	hipStream_t *stream = (hipStream_t*)malloc(sizeof(hipStream_t));
	if (!stream)
		return 0;
	hipError_t res = hipStreamCreate(stream);
	if (res == hipSuccess)
		return stream;
	else
		return 0;
}

void cuda_stream_destroy(void* stream)
{
	hipError_t res = hipStreamDestroy(*((hipStream_t*)stream));
	free(stream);
	if (res != hipSuccess)
		; //emit error
}

diy::Point* cuda_points_alloc(int size)
{
	diy::Point *ptr = 0;
	hipError_t res = hipMalloc(&ptr, size * sizeof(diy::Point));
	if (res == hipSuccess)
		return ptr;
	else
		return 0;
}

diy::Point* cuda_host_alloc(int size)
{
	diy::Point *ptr = 0;
	hipError_t res = hipHostAlloc((void**)&ptr, size * sizeof(diy::Point), hipHostMallocDefault);
	if (res == hipSuccess)
		return ptr;
	else
		return 0;
}

void cuda_points_free(diy::Point *points)
{
	hipError_t res = hipFree(points);
	if (res != hipSuccess)
		; //emit error
}

void cuda_host_free(diy::Point *points)
{
	hipError_t res = hipHostFree(points);
	if (res != hipSuccess)
		; //emit error
}

void cuda_points_to_device(const diy::Point *host_data, diy::Point *device_data, int size, void *stream)
{
	//TODO: error checking
	hipMemcpyAsync(device_data, host_data, size * sizeof(diy::Point), hipMemcpyHostToDevice, *((hipStream_t*)stream));
}

void cuda_points_from_device(const diy::Point *device_data, diy::Point *host_data, int size, void *stream)
{
	//TODO: error checking
	hipMemcpyAsync(host_data, device_data, size * sizeof(diy::Point), hipMemcpyDeviceToHost, *((hipStream_t*)stream));
}








int cuda_init_mat(
	const diy::Point *map, diy::Point *ocv_map,
	size_t rows, size_t cols, int channels, real_t finish_cx, real_t finish_cy)
{
	
	hipEvent_t start, props, maloc, kernel, fre;
hipEventCreate(&start);
hipEventCreate(&props);
hipEventCreate(&maloc);
hipEventCreate(&kernel);
hipEventCreate(&fre);

hipEventRecord(start, 0);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	int nthreads = 256;
	int nblocks = prop.multiProcessorCount * 128;

	dim3 block_dim(nthreads, 1);
	dim3 grid_dim(nblocks, 1);

	int shared_memory = nthreads * sizeof(diy::Point);
	int size = rows * cols;
	diy::Point *map_dev;
	diy::Point *ocv_map_dev;

hipEventRecord(props, 0);
hipEventSynchronize(props);
	size_t N = 2;
	size_t blocksize = (size / N) + 1;
	
	hipMalloc(&map_dev, blocksize * sizeof(diy::Point));
	hipMalloc(&ocv_map_dev, blocksize * sizeof(diy::Point));

hipEventRecord(maloc, 0);
hipEventSynchronize(maloc);

	printf("size = %d\n", size);
	for (size_t block = 0; block < N; ++block)
	{
		size_t bufsize = (block == N - 1) ? (size - block*blocksize) : blocksize;
		size_t offset = block * blocksize;
		printf("bufsize[%d] = %d\n", block, bufsize);
		printf("offset[%d] = %d\n", block, offset);

		hipMemcpy(map_dev, &map[offset], bufsize * sizeof(diy::Point), hipMemcpyHostToDevice);

		kernel_init_mat<<<nblocks, block_dim, shared_memory>>>(map_dev, ocv_map_dev, rows, cols, bufsize, finish_cx, finish_cy, offset);

		hipMemcpy(&ocv_map[offset], ocv_map_dev, bufsize * sizeof(diy::Point), hipMemcpyDeviceToHost);
	
		printf("ok\n");
	}


hipEventRecord(kernel, 0);
hipEventSynchronize(kernel);

	hipFree(map_dev);
	hipFree(ocv_map_dev);

hipEventRecord(fre, 0);
hipEventSynchronize(fre);

float time_props, time_maloc, time_kernel, time_fre, time_all;
hipEventElapsedTime(&time_props, start, props);
hipEventElapsedTime(&time_maloc, props, maloc);
hipEventElapsedTime(&time_kernel, maloc, kernel);
hipEventElapsedTime(&time_fre, kernel, fre);
hipEventElapsedTime(&time_all, start, fre);

printf("cuda time_props: %f ms\n", time_props);
printf("cuda time_maloc: %f ms\n", time_maloc);
printf("cuda time_kernel: %f ms\n", time_kernel);
printf("cuda time_fre: %f ms\n", time_fre);
printf("cuda time_all(%f): %f ms\n\n", (rows*cols*sizeof(diy::Point)) / (1024.0*1024.0), time_all);

	return 0;
}
*/
#endif
